#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

/*
 * Calculates scalar multiplication for block
 */
__global__
void ScalarMulBlock(int numElements, float* vector1, float* vector2, float *result) {
    int start_index = threadIdx.x, stride = blockDim.x;

    result[start_index] = 0;
    for (int i = start_index; i < numElements; i += stride) {
        result[index] += vector1[i] * vector2[i];
    }
}