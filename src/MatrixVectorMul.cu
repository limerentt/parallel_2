#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>


__global__ void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
    int start_index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    result[start_index] = 0;
    for (int i = start_index; i < height; i += stride) {
        for (int j = 0; j < width; ++j) {
            result[i] += matrix[i * width + j] * vector[j];
        }
    }
}