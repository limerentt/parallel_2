#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>


__global__ void KernelMatrixAdd(int height, int width, int pitch, float* A, float* B, float* result) {
  int start_index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int numElements = height * width;
  for (int i = start_index; i < numElements; i += stride) {
      int padded_index = pitch * (i / width) + i % width;
      result[padded_index] = A[padded_index] + B[padded_index];
  }
}