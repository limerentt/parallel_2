#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"


__global__ void KernelAdd(int numElements, float* x, float* y, float* result) {
    int start_index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start_index; i < numElements; i += stride) {
        y[i] += x[i];
        result[i] = y[i];
    }
}