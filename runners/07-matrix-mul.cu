#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

#include <cstdlib>
#include <iostream>

int main(int argc, char** argv) {
    int MATRIX_SIZE_CNT = std::atoi(argv[1]);
    int MATRIX_SIZE_BYTES = MATRIX_SIZE_CNT * sizeof(float);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_A = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];
    float* h_B = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];
    float* h_result = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];

    float* d_A;    
    hipMalloc(&d_A, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES);
    float* d_B;
    hipMalloc(&d_B, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES);
    float* d_result;
    hipMalloc(&d_result, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES);

    for (int i = 0; i < MATRIX_SIZE_CNT; ++i) {
        for (int j = 0; j < MATRIX_SIZE_CNT; ++j) { 
            h_A[i * MATRIX_SIZE_CNT + j] = float((i+j) % 10) / 10;
            h_B[i * MATRIX_SIZE_CNT + j] = float((i+j) % 50) / 50;
        }
    }

    hipMemcpy(d_A, h_A, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

	MatrixMul<<<dim3(1, 1), dim3(BLOCK_SIZE, BLOCK_SIZE), MATRIX_SIZE_CNT*MATRIX_SIZE_BYTES>>>(
        MATRIX_SIZE_CNT, MATRIX_SIZE_CNT, MATRIX_SIZE_CNT, d_A, d_B, d_result
    );

    hipEventRecord(stop);

    hipMemcpy(h_result, d_result, MATRIX_SIZE_CNT * MATRIX_SIZE_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(5, MATRIX_SIZE_CNT); ++i) {
        std::cout << h_result[(MATRIX_SIZE_CNT-1-i) * MATRIX_SIZE_CNT] << ' ' << h_result[MATRIX_SIZE_CNT-1-i] << std::endl;
    }

    float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time: " << milliseconds << " ms, size: " << MATRIX_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_result);

    delete [] h_A;
    delete [] h_B;
    delete [] h_result;

    return 0;
}