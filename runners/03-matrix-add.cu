#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

#include <cstdlib>
#include <iostream>

int main(int argc, char** argv) {
    int MATRIX_SIZE_CNT = std::atoi(argv[1]);
    int MATRIX_SIZE_BYTES = MATRIX_SIZE_CNT * sizeof(float);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_A = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];
    float* h_B = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];
    float* h_result = new float[MATRIX_SIZE_CNT * MATRIX_SIZE_CNT];

    size_t pitch_bytes;
    float* d_A;    
    hipMallocPitch(&d_A, &pitch_bytes, MATRIX_SIZE_BYTES, MATRIX_SIZE_BYTES);
    float* d_B;
    hipMallocPitch(&d_B, &pitch_bytes, MATRIX_SIZE_BYTES, MATRIX_SIZE_BYTES);
    float* d_result;
    hipMallocPitch(&d_result, &pitch_bytes, MATRIX_SIZE_BYTES, MATRIX_SIZE_BYTES);

    size_t pitch = pitch_bytes / sizeof(float);
    for (int i = 0; i < MATRIX_SIZE_CNT; ++i) {
        for (int j = 0; j < MATRIX_SIZE_CNT; ++j) { 
            h_A[i * MATRIX_SIZE_CNT + j] = float((i+j) % 10) / 10;
            h_B[i * MATRIX_SIZE_CNT + j] = float((i+j) % 50) / 50;
        }
    }

    hipMemcpy2D(d_A, pitch_bytes, h_A, MATRIX_SIZE_BYTES, MATRIX_SIZE_BYTES, MATRIX_SIZE_CNT, hipMemcpyHostToDevice);
    hipMemcpy2D(d_B, pitch_bytes, h_B, MATRIX_SIZE_BYTES, MATRIX_SIZE_BYTES, MATRIX_SIZE_CNT, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

	  int GRID_SIZE = (MATRIX_SIZE_CNT + BLOCK_SIZE - 1) / BLOCK_SIZE / 2 + 1;
	  KernelMatrixAdd<<<dim3(GRID_SIZE, GRID_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
        MATRIX_SIZE_CNT, MATRIX_SIZE_CNT, pitch, d_A, d_B, d_result
    );

    hipEventRecord(stop);

    hipMemcpy2D(h_result, MATRIX_SIZE_BYTES, d_result, pitch_bytes, MATRIX_SIZE_BYTES, MATRIX_SIZE_CNT, hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(5, MATRIX_SIZE_CNT); ++i) {
        std::cout << h_result[(MATRIX_SIZE_CNT-1-i) * MATRIX_SIZE_CNT] << ' ' << h_result[MATRIX_SIZE_CNT-1-i] << std::endl;
    }

    float milliseconds = 0.0;
	  hipEventElapsedTime(&milliseconds, start, stop);
	  std::cout << "Time: " << milliseconds << " ms, size: " << MATRIX_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_result);

    delete [] h_A;
    delete [] h_B;
    delete [] h_result;

    return 0;
}