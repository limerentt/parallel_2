#include "hip/hip_runtime.h"
#include "KernelMul.cuh"

#include <cstdlib>
#include <iostream>

int main(int argc, char** argv) {
    int VECTOR_SIZE_CNT = std::atoi(argv[1]);
    int VECTOR_SIZE_BYTES = VECTOR_SIZE_CNT * sizeof(float);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_x = new float[VECTOR_SIZE_CNT];
    float* h_y = new float[VECTOR_SIZE_CNT];
    float* h_result = new float[VECTOR_SIZE_CNT];

    float* d_x;
    hipMalloc(&d_x, VECTOR_SIZE_BYTES);
    float* d_y;
    hipMalloc(&d_y, VECTOR_SIZE_BYTES);
    float* d_result;
    hipMalloc(&d_result, VECTOR_SIZE_BYTES);

    for (int i = 0; i < VECTOR_SIZE_CNT; ++i) {
        h_x[i] = float(i % 10) / 5;
        h_y[i] = float(i % 50) / 10;
    }

    hipMemcpy(d_x, h_x, VECTOR_SIZE_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, VECTOR_SIZE_BYTES, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

	int GRID_SIZE = (VECTOR_SIZE_CNT + BLOCK_SIZE - 1) / BLOCK_SIZE / 2 + 1;
	KernelMul<<<GRID_SIZE, BLOCK_SIZE>>>(VECTOR_SIZE_CNT, d_x, d_y, d_result);

    hipEventRecord(stop);

    hipMemcpy(h_result, d_result, VECTOR_SIZE_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(5, VECTOR_SIZE_CNT); ++i) {
        std::cout << h_result[i] << ' ' << h_result[VECTOR_SIZE_CNT-1-i] << std::endl;
    }

    float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time: " << milliseconds << " ms, size: " << VECTOR_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    delete [] h_x;
    delete [] h_y;
    delete [] h_result;

    return 0;
}