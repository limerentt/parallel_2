#include <ScalarMulRunner.cuh>

#include <iostream>
#include <cstdlib> 

int main(int argc, char** argv) {
    int VECTOR_SIZE_CNT = std::atoi(argv[1]);
    int VECTOR_SIZE_BYTES = VECTOR_SIZE_CNT * sizeof(float);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_x = new float[VECTOR_SIZE_CNT];
    float* h_y = new float[VECTOR_SIZE_CNT];
    for (int i = 0; i < VECTOR_SIZE_CNT; ++i) {
        h_x[i] = float(i % 10) / 5;
        h_y[i] = float(i % 50) / 10;
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    float ansNonRec = ScalarMulSumPlusReduction(VECTOR_SIZE_CNT, h_x, h_y, BLOCK_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    std::cout << "ScalarMulSumPlusReduction: " << "Result: " << ansNonRec << std::endl;
    float milliseconds = 0.0;
	  hipEventElapsedTime(&milliseconds, start, stop);
	  std::cout << "ScalarMulSumPlusReduction: " << "Time: " << milliseconds << " ms, " <<
                 "size: " << VECTOR_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    hipEventRecord(start);
    float ansRec = ScalarMulTwoReductions(VECTOR_SIZE_CNT, h_x, h_y, BLOCK_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    std::cout << "ScalarMulTwoReductions: " << "Result: " << ansNonRec << std::endl;
    milliseconds = 0.0;
	  hipEventElapsedTime(&milliseconds, start, stop);
	  std::cout << "ScalarMulTwoReductions: " << "Time: " << milliseconds << " ms, " <<
                 "size: " << VECTOR_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    delete [] h_x;
    delete [] h_y;

    return 0;
}

