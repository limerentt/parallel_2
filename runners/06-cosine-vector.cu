#include "hip/hip_runtime.h"
#include <CosineVector.cuh>

#include <iostream>
#include <cstdlib> 

int main(int argc, char** argv) {
    int VECTOR_SIZE_CNT = std::atoi(argv[1]);
    int VECTOR_SIZE_BYTES = VECTOR_SIZE_CNT * sizeof(float);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_x = new float[VECTOR_SIZE_CNT];
    float* h_y = new float[VECTOR_SIZE_CNT];
    for (int i = 0; i < VECTOR_SIZE_CNT; ++i) {
        h_x[i] = float(i % 10) / 5;
        h_y[i] = float(i % 50) / 10;
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    float cos = CosineVector(VECTOR_SIZE_CNT, h_x, h_y, BLOCK_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    std::cout << "Cosinus: " << cos << std::endl;
    float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time: " << milliseconds << " ms, size: " << VECTOR_SIZE_CNT << ", block: " << BLOCK_SIZE << std::endl;

    delete [] h_x;
    delete [] h_y;

    return 0;
}