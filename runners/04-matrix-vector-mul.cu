#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

#include <cstdlib>
#include <iostream>

int main(int argc, char** argv) {
    int MATRIX_WIDTH = std::atoi(argv[1]);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float* h_matrix = new float[BLOCK_SIZE * MATRIX_WIDTH];
    float* h_vector = new float[MATRIX_WIDTH];
    float* h_result = new float[BLOCK_SIZE];

    float* d_matrix;
    hipMalloc(&d_matrix, BLOCK_SIZE * MATRIX_WIDTH * sizeof(float));
    float* d_vector;
    hipMalloc(&d_vector, MATRIX_WIDTH * sizeof(float));
    float* d_result;
    hipMalloc(&d_result, BLOCK_SIZE * sizeof(float));

    for (int i = 0; i < BLOCK_SIZE; ++i) {
        for (int j = 0; j < MATRIX_WIDTH; ++j) {
            h_matrix[i * MATRIX_WIDTH + j] = float((i+j) % 10) / 10;
        }
    }
    for (int i = 0; i < MATRIX_WIDTH; ++i) {
        h_vector[i] = float(i % 10) / 5;
    }

    hipMemcpy(d_matrix, h_matrix, BLOCK_SIZE * MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, MATRIX_WIDTH * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

	int GRID_SIZE = 1;
	MatrixVectorMul<<<GRID_SIZE, BLOCK_SIZE>>>(BLOCK_SIZE, MATRIX_WIDTH, d_matrix, d_vector, d_result);

    hipEventRecord(stop);

    hipMemcpy(h_result, d_result, BLOCK_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(5, BLOCK_SIZE); ++i) {
        std::cout << h_result[i] << ' ' << h_result[BLOCK_SIZE-1-i] << std::endl;
    }

    float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time: " << milliseconds << " ms, size: " << MATRIX_WIDTH << ", block: " << BLOCK_SIZE << std::endl;

    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);

    delete [] h_matrix;
    delete [] h_vector;
    delete [] h_result;

    return 0;
}

